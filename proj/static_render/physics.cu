#include "hip/hip_runtime.h"
#include "function/global_context.h"
#include "function/resource_manager/resource_manager.h"
#include "function/tool/fire_light_updater.h"
#include "function/type/vertex.h"
#include "physics.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#define get_bid() (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x)
#define get_tid() (get_bid() * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x)

// int vertex_cnt = extBuffers["cube1"].size / sizeof(Vertex);
// scale<<<1, vertex_cnt, 0, streamToRun>>>((Vertex*)extBuffers["cube1"].ptr);
__global__ void scale(Vertex* vertexBuffer)
{
    auto tid = get_tid();

    auto& vertex = vertexBuffer[tid];

    float scale_factor = 1.0003f;
    if (abs(vertex.pos.x) > 0.6) {
        scale_factor = 0.8f;
    }

    glm::mat4 mat = glm::mat4(1.0f);
    mat = glm::scale(mat, glm::vec3(scale_factor));
    glm::vec4 temp(vertex.pos, 1);
    temp = mat * temp;
    vertex.pos.x = temp.x;
    vertex.pos.y = temp.y;
    vertex.pos.z = temp.z;
}

// const int threadsPerBlock = 64;
// const int numBlocks = 128 * 128 * 128 / threadsPerBlock;
// fill<<<numBlocks, threadsPerBlock, 0, streamToRun>>>(
//     extImages["smoke_field"].surface_object,
//     extImages["smoke_field"].extent,
//     extImages["smoke_field"].element_size);
__global__ void fill(hipSurfaceObject_t surface_object, hipExtent extent, size_t element_size)
{
    auto id = get_tid();
    int x = id / (extent.height * extent.depth);
    int y = (id % (extent.height * extent.depth)) / extent.depth;
    int z = id % extent.depth;

    if (x >= 30 && x < 60 && y >= 20 && y < 40 && z >= 10 && z < 20) {
        surf3Dwrite(50.0f, surface_object, x * element_size, y, z);
    } else {
        surf3Dwrite(1.f, surface_object, x * element_size, y, z);
    }
}

void PhysicsEngineUser::init(Configuration& config, GlobalContext* g_ctx)
{
    CudaEngine::init(config, g_ctx);
}

void PhysicsEngineUser::initExternalMem()
{
    //     for (auto& object : g_ctx->rm->objects) {
    // #ifdef _WIN64
    //         HANDLE handle = object.getVkVertexMemHandle();
    // #else
    //         int fd = object.getVkVertexMemHandle();
    // #endif
    //
    //         const auto& mesh = g_ctx->rm->meshes[object.mesh];
    //         size_t size = mesh.data.vertices.size() * sizeof(Vertex);
    //         CudaEngine::ExtBufferDesc buffer_desc = {
    // #ifdef _WIN64
    //             handle,
    // #else
    //             fd,
    // #endif
    //             size,
    //             object.name
    //         };
    //         this->importExtBuffer(buffer_desc); // add to extBuffers internally
    //     }

    for (int i = 0; i < g_ctx->rm->fields.fields.size(); i++) {
        auto& field = g_ctx->rm->fields.fields[i];
#ifdef _WIN64
        HANDLE handle = shared_data.fields.getVkFieldMemHandle(i);
#else
        int fd = g_ctx->rm->fields.getVkFieldMemHandle(i);
#endif
        CudaEngine::ExtImageDesc image_desc = {
#ifdef _WIN64
            handle,
#else
            fd,
#endif
            128 * 256 * 128 * sizeof(float),
            sizeof(float),
            128,
            256,
            128,
            field.name
        };
        this->importExtImage(image_desc); // add to extBuffers internally
    }
}

void PhysicsEngineUser::step()
{
    waitOnSemaphore(vkUpdateSemaphore);

    if (g_ctx->rm->fields.has_temperature) {
        g_ctx->rm->fields.lights_updater->updateFireLightData(
            extImages["fire_field"].surface_object,
            streamToRun,
            g_ctx->rm->fields.lights);
    }

    signalSemaphore(cuUpdateSemaphore);
}

void PhysicsEngineUser::cleanup()
{
    CudaEngine::cleanup();
}
