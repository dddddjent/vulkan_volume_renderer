#include "hip/hip_runtime.h"
#include "core/math/math.h"
#include "core/tool/npy.hpp"
#include "fire_light_updater.h"
#include "function/type/light.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define get_bid() (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x)
#define get_tid() (get_bid() * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x)

__device__ float light_sample_gain = 0.1f;

void FireLightsUpdater::init(const FieldsConfiguration& config)
{
    sample_dim = glm::ivec3(
        config.fire_configuration.light_sample_dim[0],
        config.fire_configuration.light_sample_dim[1],
        config.fire_configuration.light_sample_dim[2]);
    sample_avg_region = glm::ivec3(
        config.fire_configuration.light_sample_avg_region[0],
        config.fire_configuration.light_sample_avg_region[1],
        config.fire_configuration.light_sample_avg_region[2]);
    for (const auto& field : config.arr) {
        if (field.name == "fire_field") {
            field_dim = arrayToVec3(field.dimension);
        }
    }
    // assert(field_dim.x % sample_dim.x == 0 && field_dim.y % sample_dim.y == 0 && field_dim.z % sample_dim.z == 0);
    sample_kernel_size = glm::ivec3(
        ceil(field_dim.x / (float)sample_dim.x),
        ceil(field_dim.y / (float)sample_dim.y),
        ceil(field_dim.z / (float)sample_dim.z));

    loadFireColorTexture(config.fire_configuration.fire_colors_path);

    hipMalloc(&d_out_intensities, sample_dim.x * sample_dim.y * sample_dim.z * sizeof(glm::vec3));
    out_intensities.resize(sample_dim.x * sample_dim.y * sample_dim.z);

    light_sample_gain = config.fire_configuration.light_sample_gain;
    hipMemcpyToSymbol(HIP_SYMBOL(light_sample_gain), &light_sample_gain, sizeof(float));
}

void FireLightsUpdater::loadFireColorTexture(const std::string& path)
{
    npy::npy_data d = npy::read_npy<float>(path);
    const auto& image_data = d.data;
    const auto& image_shape = d.shape;
    assert(image_shape.size() == 2);
    assert(image_shape[1] == 3);
    std::vector<float4> image_data4(image_data.size() / 3);
    for (int i = 0; i < image_data.size() / 3; i++) {
        image_data4[i].x = image_data[i * 3 + 0];
        image_data4[i].y = image_data[i * 3 + 1];
        image_data4[i].z = image_data[i * 3 + 2];
        image_data4[i].w = 0.0f;
    }

    hipChannelFormatDesc formatDesc = hipCreateChannelDesc<float4>();
    hipMallocArray(&fire_color_array, &formatDesc, image_shape[0], 1);
    hipMemcpy2DToArray(fire_color_array, 0, 0, image_data4.data(),
        image_shape[0] * sizeof(float4),
        image_shape[0] * sizeof(float4),
        1, hipMemcpyHostToDevice);

    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = fire_color_array;
    hipTextureDesc tex_desc;
    {
        memset(&tex_desc, 0, sizeof(tex_desc));
        tex_desc.addressMode[0] = hipAddressModeClamp;
        tex_desc.addressMode[1] = hipAddressModeClamp;
        tex_desc.filterMode = hipFilterModeLinear;
        tex_desc.readMode = hipReadModeElementType;
        tex_desc.normalizedCoords = 1;
    }
    hipCreateTextureObject(&fire_color_texture, &res_desc, &tex_desc, NULL);
}

__global__ void updateKernel(
    hipSurfaceObject_t fire_image,
    glm::ivec3 fire_image_dim,
    hipTextureObject_t fire_color_texture,
    glm::ivec3 sample_dim,
    glm::ivec3 sample_avg_region,
    glm::ivec3 sample_kernel_size,
    glm::vec3* out_intensities)
{
    auto tid = get_tid();

    auto& out = out_intensities[tid];
    int z = tid / (sample_dim.x * sample_dim.y);
    int y = (tid % (sample_dim.x * sample_dim.y)) / sample_dim.x;
    int x = tid % sample_dim.x;
    if (x >= sample_dim.x || y >= sample_dim.y || z >= sample_dim.z) {
        return;
    }
    int3 image_xyz = make_int3(
        x * sample_kernel_size.x + sample_kernel_size.x / 2,
        y * sample_kernel_size.y + sample_kernel_size.y / 2,
        z * sample_kernel_size.z + sample_kernel_size.z / 2);

    glm::dvec3 color = glm::dvec3(0.0f);
    for (int i = image_xyz.z - sample_avg_region.z; i <= image_xyz.z + sample_avg_region.z; i++) {
        for (int j = image_xyz.y - sample_avg_region.y; j <= image_xyz.y + sample_avg_region.y; j++) {
            for (int k = image_xyz.x - sample_avg_region.x; k <= image_xyz.x + sample_avg_region.x; k++) {
                auto temperature = surf3Dread<float>(
                    fire_image,
                    k * sizeof(float),
                    j,
                    i,
                    hipBoundaryModeClamp);
                float4 temp_color = tex1D<float4>(fire_color_texture, temperature);
                color += glm::vec3(temp_color.x, temp_color.y, temp_color.z);
            }
        }
    }
    color /= (2 * sample_avg_region.x + 1)
        * (2 * sample_avg_region.y + 1)
        * (2 * sample_avg_region.z + 1) / light_sample_gain;
    out = color;
}

void FireLightsUpdater::updateFireLightData(hipSurfaceObject_t fire_image, hipStream_t streamToRun, Lights& lights)
{
    dim3 thread_dim(
        std::min(sample_dim.x, 4),
        std::min(sample_dim.y, 4),
        std::min(sample_dim.z, 4));
    dim3 block_dim(
        (sample_dim.x + 3) / 4,
        (sample_dim.y + 3) / 4,
        (sample_dim.z + 3) / 4);
    updateKernel<<<block_dim, thread_dim, 0, streamToRun>>>(
        fire_image,
        field_dim,
        fire_color_texture,
        sample_dim,
        sample_avg_region,
        sample_kernel_size,
        d_out_intensities);
    hipMemcpy(
        out_intensities.data(),
        d_out_intensities,
        sample_dim.x * sample_dim.y * sample_dim.z * sizeof(float3),
        hipMemcpyDeviceToHost);

    for (int i = 0; i < sample_dim.x * sample_dim.y * sample_dim.z; i++) {
        lights.data[i].intensity = out_intensities[i];
    }
    lights.update(lights.data.data(), 0, lights.data.size());
}

void FireLightsUpdater::destroy()
{
    hipFreeArray(fire_color_array);
    hipDestroyTextureObject(fire_color_texture);
    hipFree(&d_out_intensities);
}
