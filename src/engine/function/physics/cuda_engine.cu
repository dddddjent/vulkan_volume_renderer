#include "core/config/config.h"
#include "core/vulkan/vulkan_context.h"
#include "cuda_engine.h"
#include "function/global_context.h"
#include <cassert>
// #include "shared_data.h"

void CudaEngine::importExtBuffer(const ExtBufferDesc& buffer_desc)
{
    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    {
#ifdef _WIN64
        externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
        externalMemoryDesc.handle.win32.handle = buffer_desc.handle; // File descriptor from Vulkan
#else
        externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
        externalMemoryDesc.handle.fd = buffer_desc.fd; // File descriptor from Vulkan
#endif

        externalMemoryDesc.size = buffer_desc.buffer_size;
    }
    hipExternalMemory_t ext_mem;
    hipImportExternalMemory(&ext_mem, &externalMemoryDesc);

    hipExternalMemoryBufferDesc bufferDesc = {};
    {
        bufferDesc.offset = 0;
        bufferDesc.size = buffer_desc.buffer_size;
    }
    void* dev_ptr;
    hipExternalMemoryGetMappedBuffer(&dev_ptr, ext_mem, &bufferDesc);

    extBuffers[buffer_desc.name] = {
        ext_mem,
        dev_ptr,
        buffer_desc.buffer_size
    };
}

void CudaEngine::importExtImage(const ExtImageDesc& image_desc)
{
    assert(image_desc.width * image_desc.height * image_desc.depth
        == image_desc.image_size / image_desc.element_size);

    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    {
#ifdef _WIN64
        externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
        externalMemoryDesc.handle.win32.handle = image_desc.handle; // File descriptor from Vulkan
#else
        externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
        externalMemoryDesc.handle.fd = image_desc.fd; // File descriptor from Vulkan
#endif
    }
    externalMemoryDesc.size = image_desc.image_size;
    hipExternalMemory_t ext_mem;
    hipImportExternalMemory(&ext_mem, &externalMemoryDesc);

    hipExtent extent = make_hipExtent(image_desc.width, image_desc.height, image_desc.depth);
    hipChannelFormatDesc formatDesc;
    {
        formatDesc.x = image_desc.element_size * 8;
        formatDesc.y = 0;
        formatDesc.z = 0;
        formatDesc.w = 0;
        formatDesc.f = hipChannelFormatKindFloat;
    }
    cudaExternalMemoryMipmappedArrayDesc ext_mipmapped_arr_desc;
    {
        memset(&ext_mipmapped_arr_desc, 0, sizeof(ext_mipmapped_arr_desc));
        ext_mipmapped_arr_desc.offset = 0;
        ext_mipmapped_arr_desc.formatDesc = formatDesc;
        ext_mipmapped_arr_desc.extent = extent;
        ext_mipmapped_arr_desc.flags = 0;
        ext_mipmapped_arr_desc.numLevels = 1;
    }
    hipMipmappedArray_t mipmapped_arr;
    cudaExternalMemoryGetMappedMipmappedArray(&mipmapped_arr, ext_mem, &ext_mipmapped_arr_desc);

    hipArray_t arr_0;
    hipGetMipmappedArrayLevel(&arr_0, mipmapped_arr, 0);

    hipResourceDesc res_desc = {};
    {
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = arr_0;
    }
    hipSurfaceObject_t surface_object;
    hipCreateSurfaceObject(&surface_object, &res_desc);

    extImages[image_desc.name] = {
        ext_mem,
        mipmapped_arr,
        surface_object,
        extent,
        image_desc.image_size,
        image_desc.element_size
    };
}

void CudaEngine::initExternalMem()
{
}

void CudaEngine::initSemaphore()
{
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    externalSemaphoreHandleDesc.handle.win32.handle = Vk::ctx.cuUpdateSemaphoreHandle;
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = g_ctx->vk.cuUpdateSemaphoreFd;
#endif
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(&cuUpdateSemaphore, &externalSemaphoreHandleDesc);

    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    externalSemaphoreHandleDesc.handle.win32.handle = Vk::ctx.vkUpdateSemaphoreHandle;
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = g_ctx->vk.vkUpdateSemaphoreFd;
#endif
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(&vkUpdateSemaphore, &externalSemaphoreHandleDesc);
}

void CudaEngine::init(Configuration& config, GlobalContext* g_ctx)
{
    this->g_ctx = g_ctx;

    hipStreamCreate(&streamToRun);
    initSemaphore();
    initExternalMem();
    total_frame = config.driver.total_frame;
    frame_rate = config.driver.frame_rate;
    steps_per_frame = config.driver.steps_per_frame;
    current_frame = 0;
}

void CudaEngine::step()
{
    waitOnSemaphore(vkUpdateSemaphore);

    // TODO

    signalSemaphore(cuUpdateSemaphore);
}

void CudaEngine::sync()
{
    hipDeviceSynchronize();
}

void CudaEngine::cleanup()
{
    for (auto& p : extBuffers) {
        p.second.cleanup();
    }
    for (auto& p : extImages) {
        p.second.cleanup();
    }
    hipDestroyExternalSemaphore(vkUpdateSemaphore);
    hipDestroyExternalSemaphore(cuUpdateSemaphore);
}

void CudaEngine::waitOnSemaphore(hipExternalSemaphore_t& semaphore)
{
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;
    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));
    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags = 0;

    hipWaitExternalSemaphoresAsync(
        &semaphore, &extSemaphoreWaitParams, 1, streamToRun);
}

void CudaEngine::signalSemaphore(hipExternalSemaphore_t& semaphore)
{
    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));
    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags = 0;

    hipSignalExternalSemaphoresAsync(
        &semaphore, &extSemaphoreSignalParams, 1, streamToRun);
}
